

// general
#include <stdio.h>
#include <conio.h>
#include <iostream>

// cuda
#include "hip/hip_runtime.h"


int main()
{

	int deviceNum, deviceId;
	hipGetDeviceCount(&deviceNum);
	printf("Available devices = %d pc(s)\n\n", deviceNum);

	hipDeviceProp_t prop;
	for (deviceId = 0; deviceId < deviceNum; deviceId++)
	{
		hipGetDeviceProperties(&prop, deviceId);

		// print selected device capabilities
		printf("Device properties:\n\n");
		printf("    Device id                                : %d\n", deviceId);
		printf("    Device name                              : %s\n", prop.name);
		printf("    Compute capability                       : %d.%d\n", prop.major, prop.minor);
		printf("    Total global mem                         : %llu MB\n", prop.totalGlobalMem / 1024 / 1024);
		printf("    Number of SMs                            : %d\n", prop.multiProcessorCount);
		printf("\n");
		printf("    Registers & memory \n");
		printf("        available registers per SM           : %lu k\n", prop.regsPerMultiprocessor / 1024);
		printf("        available shared mem per SM          : %llu kB\n", prop.sharedMemPerBlock / 1024);
		printf("        L2 cache size per SM                 : %d kB\n", prop.l2CacheSize / 1024);
		printf("\n");
		printf("    Thread and block data\n");
		printf("        threads per warp                     : %d\n", prop.warpSize);
		printf("        maximum threads per block            : %d\n", prop.maxThreadsPerBlock);
		printf("        maximum threads per SM               : %d\n", prop.maxThreadsPerMultiProcessor);
		printf("\n");

		// calculate number of blocks and threads
		dim3 threads(prop.warpSize, prop.maxThreadsPerBlock / prop.warpSize); // maximum number of threads per block in wapr groups
		dim3 blocks(prop.maxThreadsPerMultiProcessor * prop.multiProcessorCount / prop.maxThreadsPerBlock, 1, 1); // all block will be active! - mutex is possible

		#define threadsPerBlock (threads.x * threads.y * threads.z)
		#define totalBlocks (blocks.x * blocks.y * blocks.z)
		#define activeBlocksPerSM  (prop.maxThreadsPerMultiProcessor / threadsPerBlock)

		printf("\n");
		printf("    Settings for full occupancy and maximum sawpping  :\n\n");
		printf("\n");
		printf("        Kernel config:\n");
		printf("            total number of blocks           : %d x %d x %d = %d\n", blocks.x, blocks.y, blocks.z, totalBlocks);
		printf("            block population                 : %d cycles x %d blocks + %d blocks\n", totalBlocks / activeBlocksPerSM / prop.multiProcessorCount, activeBlocksPerSM * prop.multiProcessorCount, totalBlocks % (activeBlocksPerSM * prop.multiProcessorCount));
		printf("            SM occupancy                     : %d \n", 100 * totalBlocks / activeBlocksPerSM / prop.multiProcessorCount);
		printf("\n");
		printf("        Block data:\n");
		printf("            threads per block                : %d x %d x %d = %d\n", threads.x, threads.y, threads.z, threadsPerBlock);
		printf("            warps per block                  : %d\n", threadsPerBlock / prop.warpSize);
		printf("            maximum shared mem per block     : %llu kB\n", prop.sharedMemPerBlock / activeBlocksPerSM / 1024);
		printf("\n");
		printf("        Thread data:\n");
		printf("            average registers per thread     : %d\n", prop.regsPerBlock / activeBlocksPerSM / threadsPerBlock);
		printf("            average shared mem per thread    : %llu B\n", prop.sharedMemPerBlock / activeBlocksPerSM / threadsPerBlock);
		printf("\n");
		printf("        Others:\n");
		printf("            L1 cache transaction size        : %d B\n", 128);
		printf("            L2 cache transaction size        : %d B (only L2 = uncached loads using the generic data path)\n", 32);
		printf("            texture cache transaction size   : %d B\n", 32);
		if (prop.major <= 2) printf("            L1 cache is separated from shared mem! Use hipDeviceSetCacheConfig() !\n");
		if (prop.major >= 3) printf("            Shuffle available!\n");
		printf("\n");

	}
}

